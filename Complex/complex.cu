/*
 * complex.cu
 *
 *  Created on: Feb 1, 2015
 *      Author: yxc
 */

#ifndef COMPLEX_CU_
#define COMPLEX_CU_

#include "complex.h"

#if(path_precision == 0)
#include "complex_gd.cu"
#elif(path_precision == 1)
#include "complex_gdd.cu"
#elif(path_precision == 2)
#include "complex_gqd.cu"
#endif


#endif /* COMPLEX_CU_ */
