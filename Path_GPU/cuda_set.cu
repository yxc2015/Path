#ifndef CUDA_SET_CU_
#define CUDA_SET_CU_

#include "cuda_set.h"

void cuda_set(){
	hipSetDevice(0);
	if (hipSuccess
			!= hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte))
		std::cout << "Error!\n" << std::endl;
	else {
		std::cout << "Success!\n" << std::endl;
	}
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
}

#endif /* CUDA_SET_CU_ */
